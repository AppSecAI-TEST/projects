#include "hip/hip_runtime.h"
__global__ void dotProduct(double* output, const double* input, const int nx, const int ny)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; // Who am I?
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= ny || j >= ny || i < j)
    {
        return;
    }

    double acc = 0;
    for (int k = 0; k < nx; k++)
    {
        acc += input[k + j * nx] * input[k + i * nx];
    }

    output[i + j * ny] = acc;
}

void make_stuff(...---...)
{
    double* inputCPU = 0;
    double* outputCPU = 0;
    hipHostMalloc((void**) &inputCPU, ny * nx * sizeof (double));
    hipHostMalloc((void**) &outputCPU, ny * ny * sizeof (double));

    double* inputGPU = 0;
    double* outputGPU = 0;
    hipMalloc((void**) &inputGPU, ny * nx * sizeof (double));
    hipMalloc((void**) &outputGPU, ny * ny * sizeof (double));

    int i, j;
    double acc, mean, tmp, norm;

    for (j = 0; j < ny; j++)
    {
        acc = 0;
        for (i = 0; i < nx; i++)
        {
            acc += data[i + j * nx];
        }
        mean = acc / (double) nx;
        acc = 0;
        for (i = 0; i < nx; i++)
        {
            tmp = data[i + j * nx] - mean;
            inputCPU[i + j * nx] = tmp;
            acc += pow(tmp, 2);
        }
        norm = sqrt(acc);
        for (i = 0; i < nx; i++)
        {
            inputCPU[i + j * nx] /= norm;
        }
    }

    hipMemcpy(inputGPU, inputCPU, ny * nx * sizeof (double), hipMemcpyHostToDevice);
    dim3 dimBlock(8, 8);
    dim3 dimGrid((ny + dimBlock.y - 1) / dimBlock.y, (ny + dimBlock.y - 1) / dimBlock.y);

    dotProduct << <dimGrid, dimBlock>>>(outputGPU, inputGPU, nx, ny);
    hipDeviceSynchronize();

    hipMemcpy(outputCPU, outputGPU, ny * ny * sizeof (double), hipMemcpyDeviceToHost);

    for (int i = 0; i < ny; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            result[j + i * ny] = outputCPU[j + i * ny];
        }
    }

    hipHostFree(inputCPU);
    hipHostFree(outputCPU);
    hipFree(inputGPU);
    hipFree(outputGPU);
}

